#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "func_cuda.h"

#ifndef PROBDIM
#define PROBDIM 2
#endif

#define INF 1e99

/* IDEA : Calculate a matrix of size [QUERYELEMS, TRAINELEMS] where the element [i,j] is the
 * distance between the i-th query point and the j-th training point. Since this is a lightweight
 * task, it may be assigned to a device thread. 
 * Then we may find the k minimum values of each row. For each one of those k values of a row, 
 * we will be preserving their index inside the matrix and thus the index of the point inside the TRAINELEMS array.
 * We may then predict the value of the query and calculate the errors, accumulating them in a device array
 * copyout those arrays to host, reduce them to the desired metrics and exit.
 */

// TODO : Generalize this
// number of rows (in the overall matrix) to calculate in parallel
#define QUERY_BLOCK_SIZE 		16

// Each thread block should calculate a tile of this "global" matrix. Since the max size of a thread block is 1024 we
// derive the TRAIN_BLOCK_SIZE (which is the number of columns (in the overall matrix) to calculate in parallel)
// in the following manner.
#define TRAIN_BLOCK_SIZE 		(1024 / QUERY_BLOCK_SIZE)

// The amount of tiles (thread blocks) required to calculate a full block row of the matrix
#define ROW_THREAD_BLOCKS 		(TRAINELEMS / TRAIN_BLOCK_SIZE)

__device__ void thread_block_reduction(double *dist_vec, int *global_nn_idx, double *global_nn_dist, int query_idx, int k)
{	
        // One thread uses a single buf element (1 to 1 correspondance)
        __shared__ int buf[QUERY_BLOCK_SIZE * TRAIN_BLOCK_SIZE];
        double next_dist;

	for(int neigh = 0; neigh < k; neigh++) // all threads will participate in finding each of the k neighbors
        {
                // Initially the buffer line for each query (indexed by threadIdx.y) ([0, QUERY_BLOCK_SIZE-1])
                // should contain the indexes of the local training point (threadIdx.x) ([0, TRAIN_BLOCK_SIZE-1])
                // since the minimum distance it knows yet is the one that this thread calculated.
                // Thus, on the left handside, we index the buffer cell that corresponds to the current thread
                // and on the right handside we store the index of the local training point.

                // Because the training point evaluated at each column does not change, on the right handside we
                // may also store the position of the thread, inside the thread block. We may later recover the threadIdx.x
                // (i.e. the index of the local training point), by subtracting the offset introduced by the local query (i.e. threadIdx.y * blockDim.x).
                // We do this last step so we may easily index the dist_vec array, without having to calculate the offset indroduced by the local query
                // at each iteration.
                buf[threadIdx.y * blockDim.x + threadIdx.x] = threadIdx.y * blockDim.x + threadIdx.x;

                for (int j = 0; j < (int)log2f(blockDim.x); j++) // reduce the number of threads
                {
                /*      __syncthreads();
                        // TODO: Replace mod with something more efficient (?)
                        if (threadIdx.x % (int)pow(2, j+1) == 0)
                        {
                                int curr_idx = threadIdx.y * blockDim.x + threadIdx.x;
                                int next_idx = curr_idx + pow(2, j);
                                buf[curr_idx] = (dist_vec[buf[curr_idx]] < dist_vec[buf[next_idx]]) ? curr_idx : next_idx;
                        } 
                */
                        int curr_idx = threadIdx.y * blockDim.x + threadIdx.x;
                        int next_idx = curr_idx + pow(2, j);
			next_idx = next_idx > blockDim.x - 1 ? 0 : next_idx;
                        next_dist = dist_vec[buf[next_idx]];
                        __syncthreads();
                        buf[curr_idx] = (dist_vec[buf[curr_idx]] < next_dist) ? curr_idx : next_idx;
                }
                
                // Only threads with threadIdx.x == 0 does the last reduction, thus there is no need to __synchthreads here.
		if (threadIdx.x == 0)
		{
			// Recover threadIdx.x as explained above
			// global_nn_idx[(blockIdx.x*blockDim.y + threadIdx.y)*k + neigh] =
			// 		blockIdx.x * blockDim.x + (buf[threadIdx.y * blockDim.x + 0] - threadIdx.y * blockDim.x);
					
			// global_nn_dist[(blockIdx.x*blockDim.y + threadIdx.y)*k + neigh] =
			// 		dist_vec[buf[threadIdx.y * blockDim.x + 0]];

                        global_nn_idx[(threadIdx.y*gridDim.x + blockIdx.x)*k + neigh] = 
                                        blockIdx.x * blockDim.x + (buf[threadIdx.y * blockDim.x + 0] - threadIdx.y * blockDim.x);

                        global_nn_dist[(threadIdx.y*gridDim.x + blockIdx.x)*k + neigh] = 
                                        dist_vec[buf[threadIdx.y * blockDim.x + 0]];

			dist_vec[buf[threadIdx.y * blockDim.x + 0]] = INF;
		}      
        }
}

__global__ void compute_distances_kernel(double *mem, double *query_mem, int query_block_offset, int query_block_size,
					 int *global_nn_idx, double *global_nn_dist, int k, int dim,
					 size_t trainel_block_size, size_t queryel_block_size)
{
	extern __shared__ char arr[];
	
	double *trainel_block = (double *)arr;
	double *query_block = (double *)(arr + trainel_block_size);
	double *dist_vec = (double *)(arr + trainel_block_size + queryel_block_size);

	int trainel_block_offset = blockIdx.x * blockDim.x;

	// same thread-block row threads -> same threadIdx.y -> compute with the same query (global ty)
	
        // Indexes for the global memory (be careful you only have the partial matrix in memory)
        int global_tx = blockIdx.x * blockDim.x + threadIdx.x; // matrix col this thread is in
        int global_ty = blockIdx.y * blockDim.y + threadIdx.y; // matrix row this thread is in
	
        // Indexes for the shared memory
	int local_trainel_idx = threadIdx.x; // thread-block col of current thread
	int local_query_idx = threadIdx.y; // thread-block row of current thread

	// load data into the proper shared memory regions from device global memory
	// only the thread block's "zero" thread loads these data
        if(threadIdx.x == 0 && threadIdx.y == 0) 
        {
		memcpy(trainel_block, mem + trainel_block_offset, trainel_block_size);

		/* __CHANGE__: We had forgotten to use the query_block_offset argument passed,
		 * for the calculation of the query_mem "loading starting point".
		 */
		memcpy(query_block, query_mem + query_block_offset, queryel_block_size);
        }
        
	__syncthreads();

	// each thread computes the distance for its query point with its training element
	// then it updates its respective position in the distances vector

	/* __CHANGE__: Prior to this change each thread called compute_dist with
	 * &query_block[local_query_idx] and &trainel_block[local_trainel_idx],
	 * which is WRONG.
	 * For example, ff a thread needs to compute the distance of query point 1 and
	 * training element 1, we must not index the respective shared memory arrays,
	 * using 1, because those arrays are 1D arrays whose elements are doubles,
	 * but we must think of them as **"vectors" of size dim**.
	 * So, when a thread needs the training element "1" it does not need
	 * trainel_block[1] but trainel_block[1*dim], to skip the previous vector(s).
	 */
	dist_vec[local_query_idx * blockDim.x + local_trainel_idx] = 
                	compute_dist(&query_block[local_query_idx * dim], &trainel_block[local_trainel_idx * dim], dim);

	__syncthreads();

        thread_block_reduction(dist_vec, global_nn_idx, global_nn_dist, local_query_idx, k);
}

__global__ void reduce_distance_kernel(int *global_nn_idx, double *global_nn_dist, int len, int k, size_t dist_vec_size)
{
        __shared__ int buf[QUERY_BLOCK_SIZE * TRAIN_BLOCK_SIZE]; // static shared memory buffer
	
        extern __shared__ char shared_arr[]; 			 // dynamically allocated shared memory
        double *dist_vec = (double *)shared_arr;
        int *idx_vec = (int *)(shared_arr + dist_vec_size);

        // Indexes for the global memory (be careful you only have the partial matrix in memory)
        int global_tx = blockIdx.x * blockDim.x + threadIdx.x; // matrix col this thread is in
        int global_ty = blockIdx.y * blockDim.y + threadIdx.y; // matrix row this thread is in
	
        // Indexes for the shared memory
	int local_neigh_idx = threadIdx.x; // thread-block col of current thread -> local "candidate neighbor"
	int local_query_idx = threadIdx.y; // thread-block row of current thread -> local query point under reduction (same as global)

	// load data into the proper shared memory regions from device global memory
	// only the thread block's "zero" thread loads these data
        if(threadIdx.x == 0) 
        {
		memcpy(dist_vec + threadIdx.y * len, global_nn_dist + (threadIdx.y * gridDim.x + blockIdx.x)*k, len * sizeof(double));
		memcpy(idx_vec + threadIdx.y * len, global_nn_idx + (threadIdx.y * gridDim.x + blockIdx.x)*k, len * sizeof(int));
        }

	__syncthreads();

        // Decide on the number of iterations required to do the reduction to len elements
	int num_iter = log2f(len);

        int thread_block_local_tid = threadIdx.y * blockDim.x  + threadIdx.x;
	for(int neigh = 0; neigh < k; neigh++)
	{
                // first iteration will be performed on data twice the size of the thread block
                // so we may use all threads for this first step, we should only be careful with the indexing
                // If the length (len) of the data we want to reduce is less than the available number of threads for
                // each query (blockDim.x), we obviously should not perform this step
                if (len > blockDim.x)
                {
		        __syncthreads();
                        buf[thread_block_local_tid] = (dist_vec[2*thread_block_local_tid] < dist_vec[2*thread_block_local_tid+1]) 
                                		      ? 2 * thread_block_local_tid
                                		      : 2 * thread_block_local_tid + 1;
                        --num_iter;
                }
		
                buf[thread_block_local_tid] = threadIdx.y * blockDim.x  + threadIdx.x; // = thread_block_local_tid

                for (int j = 0; j < num_iter; j++) // reduce the number of threads
                {
                        __syncthreads();
                        // TODO: Replace mod with something more efficient (?)
                        if (threadIdx.x % (int)pow(2, j+1) == 0 && threadIdx.x < len)
                        {
                                int curr_idx = thread_block_local_tid;
                                int next_idx = curr_idx + pow(2, j);
                                buf[curr_idx] = (dist_vec[buf[curr_idx]] < dist_vec[buf[next_idx]]) ? curr_idx : next_idx;
                        }
                }

                // Only threads with threadIdx.x == 0 does the last reduction, thus there is no need to __synchthreads here.
                if (threadIdx.x == 0)
                {
                        // TODO: Recover the threadIdx.x from buf[0] (for each query point)

                        // Store the best k nearest neighbors calculated by this thread to the global 3d tensor
                        // of size [QUERY_BLOCK_SIZE, gridDim.x, k].
                        // As a reminder we may think about the current size of nn_dist and nn_idx,
                        // which is [QUERY_BLOCK_SIZE, (len * gridDim.x) / k, k], 
                        // where (len * gridDim.x) / k -> the number of thread blocks we used on the previous iteration.
                        // len * gridDim.x -> the number of elements stored on the previous call of this kernel
                        // (len * gridDim.x) / k -> number of thread blocks that wrote those elements on the previous kernel call, 
                        // since each wrote k neighbors
                        
			global_nn_dist[(threadIdx.y * gridDim.x + blockIdx.x) * k + neigh] =
                                                dist_vec[buf[threadIdx.y * blockDim.x + 0]];
			
			global_nn_idx[(threadIdx.y * gridDim.x + blockIdx.x) * k + neigh] =  
                                                blockIdx.x * blockDim.x + (buf[threadIdx.y * blockDim.x + 0] - threadIdx.y * blockDim.x);
			
			dist_vec[buf[threadIdx.y * blockDim.x + 0]] = INF;
                }
	}
}

__global__ void predict_query_values(double *dev_ydata, double *dev_query_ydata, int *dev_nn_idx, int query_block_start, int k, double *dev_sse, double *dev_err)
{
	double neigh_vals[NNBS];

	// if(tid <)
        // each thread runs for a query (thus the global thread id is equal to the query id inside the quey block)
	int tid = threadIdx.x; // running with a 1D Thread Block
        int query_idx = query_block_start + tid;
	for(int i = 0; i < k; i++)
		neigh_vals[i] = dev_ydata[dev_nn_idx[tid * k + i]];
                
	// call predict_value
	double yp = predict_value(neigh_vals, k);

	// compute error metrics
	dev_sse[query_idx] = (dev_query_ydata[query_idx] - yp) * (dev_query_ydata[query_idx] - yp);
        dev_err[query_idx] = 100.0 * fabs((yp - dev_query_ydata[query_idx]) / dev_query_ydata[query_idx]);
}

int main(int argc, char **argv)
{
	/* Load all data from files in memory */
	if (argc != 3)
	{
		printf("usage: %s <trainfile> <queryfile>\n", argv[0]);
		exit(1);
	}
	char *trainfile = argv[1];
	char *queryfile = argv[2];

        assert(TRAIN_BLOCK_SIZE > 32);

	// int dev;
	// hipGetDevice(&dev); // get GPU device number
	// hipDeviceProp_t prop;
	// hipGetDeviceProperties(&prop, dev); // get cuda Device Properties
	// size_t shared_mem_size = prop.sharedMemPerBlock; // shared memory size

#if defined(DEBUG)
        /* Create/Open an output file */
        FILE *fpout = fopen("output.knn.txt","w");
#endif
        int vector_size = PROBDIM + 1;
	double *dev_mem, *dev_ydata, *dev_query_ydata, *dev_query_mem, *dev_nn_dist, *dev_sse, *dev_err;
	int *dev_nn_idx;
	
	// ******************************************************************
	// ************************** Host mallocs **************************
	// ******************************************************************
	double *mem = (double *)malloc(TRAINELEMS * vector_size * sizeof(double));	 // Training Element vectors
	double *ydata = (double *)malloc(TRAINELEMS * sizeof(double));	  	  	 // Training Element Surrogate values
	double *query_mem = (double *)malloc(QUERYELEMS * vector_size * sizeof(double)); // Query Element vectors
	double *query_ydata = (double*)malloc(QUERYELEMS * sizeof(double));		 // Query Element Surrogate values

	double *train_buf = (double*)malloc(TRAINELEMS * PROBDIM * sizeof(double));
	double *query_buf = (double*)malloc(QUERYELEMS * PROBDIM * sizeof(double));

	// ******************************************************************
	// ************************** Load Data *****************************
	// ******************************************************************
	load_binary_data(trainfile, mem, TRAINELEMS * (PROBDIM + 1));
	load_binary_data(queryfile, query_mem, QUERYELEMS * vector_size);

	extract_vectors(mem, train_buf, TRAINELEMS, PROBDIM + 1, PROBDIM);

	// construct a "pure" query elements array to pass to the device
        // TODO: Have the same in/out buffer -> overwriting
	extract_vectors(query_mem, query_buf, QUERYELEMS, PROBDIM + 1, PROBDIM);

	// ******************************************************************
	// ************************** Device mallocs ************************
	// ******************************************************************
	// allocate global memory on the device for the training element and query vectors

        hipMalloc((void**)&dev_mem, TRAINELEMS * PROBDIM * sizeof(double));	 	 // Device Training Element vectors
        hipMalloc((void**)&dev_ydata, TRAINELEMS * sizeof(double));			 // Device Training Element Surrogate values

	hipMalloc((void**)&dev_query_mem, QUERYELEMS * PROBDIM * sizeof(double)); 	 // array to host "pure" query element vectors
        hipMalloc((void**)&dev_query_ydata, QUERYELEMS * sizeof(double));		 // Device Query Element Surrogate values

	/* Allocate enough space for each thread block to store the k nearest neighbors it found (partial reduction results)
	 * We must preserve both the index of the nearest neighbors and their distance from the query point
	 * since we will later reduce the k*ROW_THREAD_BLOCKS nearest neighbors to the final k for each query point.
	 */
	hipMalloc((void**)&dev_nn_dist, ROW_THREAD_BLOCKS * (QUERY_BLOCK_SIZE * NNBS) * sizeof(double));
	hipMalloc((void**)&dev_nn_idx, ROW_THREAD_BLOCKS * (QUERY_BLOCK_SIZE * NNBS) * sizeof(int));

	hipMalloc((void **)&dev_sse, QUERYELEMS * sizeof(double));
	hipMalloc((void **)&dev_err, QUERYELEMS * sizeof(double));

	hipMemset(dev_sse, 0, QUERYELEMS * sizeof(double));
	hipMemset(dev_err, 0, QUERYELEMS * sizeof(double));

	// ******************************************************************
	// ************************** Host data init ************************
	// ******************************************************************
        // init all data on CPU **Then** send them to GPU

	for (int i = 0; i < TRAINELEMS; i++) // init training elements' surrogate values
	{
#if defined(SURROGATES)
		ydata[i] = mem[i * vector_size + PROBDIM];
#else
		ydata[i] = 0;
#endif
	}

	for (int i = 0; i < QUERYELEMS; i++) // init query elements' surrogate values
	{
#if defined(SURROGATES)
		query_ydata[i] = query_mem[i * vector_size + PROBDIM];		
#else
		query_ydata[i] = 0;
#endif
	}

	// ******************************************************************
	// ************************** Copyout data to device ****************
	// ******************************************************************	
	hipMemcpy(dev_mem, train_buf, TRAINELEMS * PROBDIM * sizeof(double), hipMemcpyHostToDevice); 		 // copy train elems
	hipMemcpy(dev_ydata, ydata, TRAINELEMS * sizeof(double), hipMemcpyHostToDevice); 			 // copy train elems surrogate values
       
	hipMemcpy(dev_query_mem, query_buf, QUERYELEMS * PROBDIM * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_query_ydata, query_ydata, QUERYELEMS * sizeof(double), hipMemcpyHostToDevice); 		 // copy query elems surrogate values

	dim3 block_size(TRAIN_BLOCK_SIZE, QUERY_BLOCK_SIZE, 1);
	dim3 grid_dim(ROW_THREAD_BLOCKS, 1, 1);

	size_t dist_vector_size = QUERY_BLOCK_SIZE * TRAIN_BLOCK_SIZE * sizeof(double);
	size_t trainel_block_size = TRAIN_BLOCK_SIZE * PROBDIM * sizeof(double);
	size_t queryel_block_size = QUERY_BLOCK_SIZE * PROBDIM * sizeof(double);

	size_t shared_mem_size = dist_vector_size + trainel_block_size + queryel_block_size;

	/* Each thread block's shared memory is comprised of :
	 * 
	 *   - a trainel_block -> TRAIN_BLOCK_SIZE * PROBDIM * sizeof(double)
	 *   
	 *   - a queryel_block -> QUERY_BLOCK_SIZE * PROBDIM * sizeof(double)
	 *
	 *   - a dist_vector -> QUERY_BLOCK_SIZE * TRAIN_BLOCK_SIZE * sizeof(double)
	 */

	assert(block_size.x % 2 == 0);
	float num_thread_blocks;
        int len;
        size_t reduction_shared_mem_size;

	/* COMPUTATION PART */
        double t_start = gettime();
	
	for(int i = 0; i < QUERYELEMS; i += QUERY_BLOCK_SIZE)
	{
        	compute_distances_kernel<<<grid_dim, block_size, shared_mem_size>>>(dev_mem, dev_query_mem, i, QUERY_BLOCK_SIZE, 
		 					dev_nn_idx, dev_nn_dist, NNBS, PROBDIM, trainel_block_size, queryel_block_size);  // compute a "chunk" of rows
                // Check for any cuda errors you might be missing
                // printf("compute_distances_kernel error code: %d\n", hipGetLastError());
                assert(hipGetLastError() == 0);
	        
		num_thread_blocks = ROW_THREAD_BLOCKS * NNBS / (2 * block_size.x);
                dim3 reduction_block_size(block_size.x, QUERY_BLOCK_SIZE, 1);
		while(1)
		{                        
                        // Calculate the number of thread blocks required
			dim3 reduction_grid_dim((int)num_thread_blocks, 1, 1);
			
                        // Calculate the amount of elements in each thread block (per query)
                        len = num_thread_blocks * (2 * block_size.x) / ceil(num_thread_blocks);
                        // Fix the num_thread_blocks, so if there are less than block_size.x elements left
                        // you will still use 1 thread block in order to reduce them to NNBS neighbors
			num_thread_blocks = ceil(num_thread_blocks);

                        // For each query we will load in the shared memory 2*block_size.x nearest neighbors (i.e. their distances and idx)
                        // The shared memory will be structured as:
                        // [query0 distances][query1 distances]...[query0 indexes][query1 indexes]...
                        // Calculate the shared memory size
                        size_t reduction_dist_vector_size = QUERY_BLOCK_SIZE * len * sizeof(double); 
                        size_t reduction_idx_vector_size = QUERY_BLOCK_SIZE * len * sizeof(int);

                        // size_t reduction_dist_vector_size = QUERY_BLOCK_SIZE * 2 * block_size.x * sizeof(double); // initial thought
                        // size_t reduction_idx_vector_size = QUERY_BLOCK_SIZE * 2 * block_size.x * sizeof(int);
                        reduction_shared_mem_size = reduction_dist_vector_size + reduction_idx_vector_size;
			reduce_distance_kernel<<<reduction_grid_dim, reduction_block_size, reduction_shared_mem_size>>>
                                (dev_nn_idx, dev_nn_dist, len, NNBS, reduction_dist_vector_size);
                        // Check for any cuda errors you might be missing
                        // printf("reduce_distance_kernel error code: %d\n", hipGetLastError());
                        assert(hipGetLastError() == 0);
			
                        // update control variable
                        if (num_thread_blocks == 1)
                                break;
			num_thread_blocks = num_thread_blocks * NNBS / (2 * block_size.x);
		}

                // Find yp for each query and error metrics
		predict_query_values<<<1, QUERY_BLOCK_SIZE>>>(dev_ydata, dev_query_ydata, dev_nn_idx, i, NNBS, dev_sse, dev_err);
                // Check for any cuda errors you might be missing
                assert(hipGetLastError() == 0);
	}

	// sync device and host before getting final time
	hipDeviceSynchronize();
        double t_sum = gettime() - t_start;

	double sse = 0.0f, err_sum = 0.0f;
	double *buf = (double *)malloc(QUERYELEMS * sizeof(double));
	hipMemcpy(buf, dev_sse, QUERYELEMS * sizeof(double), hipMemcpyDeviceToHost);
	for (int i = 0; i < QUERYELEMS; i++)
		sse += buf[i];

	hipMemcpy(buf, dev_err, QUERYELEMS * sizeof(double), hipMemcpyDeviceToHost);
	for (int i = 0; i < QUERYELEMS; i++)
		err_sum += buf[i];

	/* CALCULATE AND DISPLAY RESULTS */
	// these will be calculated on the CPU
	double mse = sse / QUERYELEMS;
	double ymean = compute_mean(query_ydata, QUERYELEMS);
	double var = compute_var(query_ydata, QUERYELEMS, ymean);
	double r2 = 1 - (mse / var);

	printf("Results for %d query points\n", QUERYELEMS);
	printf("APE = %.2f %%\n", err_sum / QUERYELEMS);
	printf("MSE = %.6f\n", mse);
	printf("R2 = 1 - (MSE/Var) = %.6lf\n", r2);

	printf("Total time = %lf secs\n", t_sum);
	// printf("Time for 1st query = %lf secs\n", t_first);
	// printf("Time for 2..N queries = %lf secs\n", t_sum - t_first);
	// printf("Average time/query = %lf secs\n", (t_sum - t_first) / (QUERYELEMS - 1));

	/* CLEANUP */

#if defined(DEBUG)
	/* Close the output file */
	fclose(fpout);
#endif

	free(mem);
	free(ydata);
	free(query_mem);
	free(query_ydata);
	free(buf);
        free(train_buf);
	free(query_buf);

        hipFree(dev_mem);
	hipFree(dev_ydata);
	hipFree(dev_query_mem);
        hipFree(dev_query_ydata);
	hipFree(dev_nn_dist);
	hipFree(dev_nn_idx);
	hipFree(dev_sse);
	hipFree(dev_err);

	return 0;
}
