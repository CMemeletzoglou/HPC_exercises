#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "func_cuda.h"

#ifndef PROBDIM
#define PROBDIM 2
#endif

#define TRAIN_BLOCK_SIZE 256 //initial testing

static int ntrain_blocks, train_block_size;

// __global__ void knn_kernel(double *mem, double *ydata, double *query_mem, query_t *query_ydata, int k, double *sse, double *err, int *nblocks, int *block_size)
__global__ void knn_kernel(double *mem, double *ydata, double *query_mem, double *query_ydata, int k, double *sse, double *err, int *nblocks, int *block_size)
{
	double t0, t1, t_first = 0.0, t_sum = 0.0;
        double yp;
	
	int tx = blockIdx.x * blockDim.x + threadIdx.x;

	int *thread_block_nn_idx;
	hipMalloc((void **)&thread_block_nn_idx, blockDim.x * blockDim.y * k * sizeof(int));

	// threads of the same column, will handle the same query
	// threads of the same row will compute the neighbors of the query assigned to their column, 
	// for the training element block assigned to their row

	// so tx indicates the query element index, while ty indicates the training element block index 
	
	if(tx < QUERYELEMS && ty < *nblocks) // stay inside problem boundaries
		compute_knn_brute_force_cuda(mem, ydata, &(query_mem[tx]), thread_block_nn_idx, PROBDIM, k, ty, *block_size);


		// compute_knn_brute_force_cuda(dev_mem, dev_ydata, &(dev_queries[tx]), PROBDIM, k, ty, *block_size);






	// for (int block_start = 0; block_start < TRAINELEMS; block_start += TRAIN_BLOCK_SIZE)
	// 	compute_knn_brute_force_cuda(dev_mem, dev_ydata, &(dev_queries[tid]), PROBDIM, k, block_start, TRAIN_BLOCK_SIZE); // brute-force / linear search

	// Predict query point value
        // yp = predict_value(&(dev_queries[tid]), dev_ydata, PROBDIM, NNBS);
        // sse[tid] = (dev_query_ydata[tid] - yp) * (dev_query_ydata[tid] - yp);
        // err[tid] = 100.0 * fabs((yp - dev_query_ydata[tid]) / dev_query_ydata[tid]);

	hipFree(thread_block_nn_idx);
}

__global__ void compute_dist_kernel(double *mem, double *query_mem)
{
        extern __shared__ double arr[];
        double *train_elems = arr;
        double *dists_vec = arr + TRAIN_BLOCK_SIZE * PROBDIM * sizeof(double);
        double *query_vec = dists_vec + TRAIN_BLOCK_SIZE * sizeof(double);

        
}

int main(int argc, char *argv[])
{
	/* Load all data from files in memory */
	if (argc != 3)
	{
		printf("usage: %s <trainfile> <queryfile>\n", argv[0]);
		exit(1);
	}
	char *trainfile = argv[1];
	char *queryfile = argv[2];

	// int dev;
	// hipGetDevice(&dev); // get GPU device number
	// hipDeviceProp_t prop;
	// hipGetDeviceProperties(&prop, dev); // get cuda Device Properties
	// size_t shared_mem_size = prop.sharedMemPerBlock; // shared memory size

#if defined(DEBUG)
        /* Create/Open an output file */
        FILE *fpout = fopen("output.knn.txt","w");
#endif
        int vector_size = PROBDIM + 1;
	double *dev_mem, *dev_ydata, *dev_query_ydata, *dev_sse, *dev_err;

	double *dev_query_mem;
	
	// ******************************************************************
	// ************************** Host mallocs **************************
	// ******************************************************************
	double *mem = (double *)malloc(TRAINELEMS * vector_size * sizeof(double));	 // Training Element vectors
	double *ydata = (double *)malloc(TRAINELEMS * sizeof(double));	  	  	 // Training Element Surrogate values
	double *query_mem = (double *)malloc(QUERYELEMS * vector_size * sizeof(double)); // Query Element vectors
	double *query_ydata = (double*)malloc(QUERYELEMS * sizeof(double));		 // Query Element Surrogate values

	double *train_buf = (double*)malloc(TRAINELEMS * PROBDIM * sizeof(double));
	double *query_buf = (double*)malloc(QUERYELEMS * PROBDIM * sizeof(double));

	// ******************************************************************
	// ************************** Load Data *****************************
	// ******************************************************************
	load_binary_data(trainfile, mem, NULL, TRAINELEMS * (PROBDIM + 1));
	load_binary_data(queryfile, query_mem, NULL, QUERYELEMS * vector_size);

	extract_vectors(mem, train_buf, TRAINELEMS, PROBDIM + 1, PROBDIM);

	// construct a "pure" query elements array to pass to the device
	extract_vectors(query_mem, query_buf, QUERYELEMS, PROBDIM + 1, PROBDIM);

	// ******************************************************************
	// ************************** Device mallocs ************************
	// ******************************************************************
	// allocate global memory on the device for the training element and query vectors

        hipMalloc((void**)&dev_mem, TRAINELEMS * PROBDIM * sizeof(double));	 	 // Device Training Element vectors
        hipMalloc((void**)&dev_ydata, TRAINELEMS * sizeof(double));			 // Device Training Element Surrogate values

        hipMalloc((void**)&dev_query_ydata, QUERYELEMS * sizeof(double));		 // Device Query Element Surrogate values

	hipMalloc((void**)&dev_query_mem, QUERYELEMS * PROBDIM * sizeof(double)); // array to host "pure" query element vectors

	hipMalloc((void **)&dev_sse, QUERYELEMS * sizeof(double));
	hipMalloc((void **)&dev_err, QUERYELEMS * sizeof(double));

	hipMemset(dev_sse, 0, QUERYELEMS * sizeof(double));
	hipMemset(dev_err, 0, QUERYELEMS * sizeof(double));

	// ******************************************************************
	// ************************** Host data init ************************
	// ******************************************************************
        // init all data on CPU **Then** send them to GPU

	for (int i = 0; i < TRAINELEMS; i++) // init training elements' surrogate values
	{
#if defined(SURROGATES)
		ydata[i] = mem[i * vector_size + PROBDIM];
#else
		ydata[i] = 0;
#endif
	}

	for (int i = 0; i < QUERYELEMS; i++) // init query elements' surrogate values
	{
#if defined(SURROGATES)
		query_ydata[i] = query_mem[i * vector_size + PROBDIM];		
#else
		query_ydata[i] = 0;
#endif
	}

	// ******************************************************************
	// ************************** Copyout data to device ****************
	// ******************************************************************	
	hipMemcpy(dev_mem, train_buf, TRAINELEMS * PROBDIM * sizeof(double), hipMemcpyHostToDevice); 		 // copy train elems
	hipMemcpy(dev_ydata, ydata, TRAINELEMS * sizeof(double), hipMemcpyHostToDevice); 			 // copy train elems surrogate values
       
	hipMemcpy(dev_query_ydata, query_ydata, QUERYELEMS * sizeof(double), hipMemcpyHostToDevice); 		 // copy query elems surrogate values

	hipMemcpy(dev_query_mem, query_buf, QUERYELEMS * PROBDIM * sizeof(double), hipMemcpyHostToDevice);

	/* Each thread block will have #rows = #Training Element blocks and #cols = some multiple of 32.
	 * Assume that we divide the Training Elements into 2^5 = 32 Training Element blocks, where
	 * each block contains 2^15 Training Elements
	 */
	
	ntrain_blocks = TRAINELEMS / TRAIN_BLOCK_SIZE; // 2^12

        size_t shared_mem_size = ((TRAIN_BLOCK_SIZE * PROBDIM) + TRAIN_BLOCK_SIZE + PROBDIM) * sizeof(double);

        // int *dev_train_block_size, *dev_ntrain_blocks;
	// hipMalloc((void**)&dev_train_block_size, sizeof(int));
	// hipMalloc((void**)&dev_ntrain_blocks, sizeof(int));
	// hipMemcpy(&dev_train_block_size, &train_block_size, sizeof(int), hipMemcpyHostToDevice);
	// hipMemcpy(&dev_ntrain_blocks, &ntrain_blocks, sizeof(int), hipMemcpyHostToDevice);

	/* COMPUTATION PART */
        double t_start = gettime();

        compute_dist_kernel<<<ntrain_blocks, train_block_size, shared_mem_size>>>(dev_mem, query_mem);

        hipDeviceSynchronize(); // wait before calling knn kernel

        // knn_kernel<<<ntrain_blocks, train_block_size, shared_mem_size>>>(dev_mem, dev_ydata, dev_query_mem, dev_query_ydata, NNBS, dev_sse, dev_err, dev_ntrain_blocks, dev_train_block_size);
	

	// sync device and host before getting final time
	hipDeviceSynchronize();
        double t_sum = gettime() - t_start;

	double sse = 0.0f, err_sum = 0.0f;
	double *buf = (double *)malloc(QUERYELEMS * sizeof(double));
	hipMemcpy(buf, dev_sse, QUERYELEMS * sizeof(double), hipMemcpyDeviceToHost);
	for (int i = 0; i < QUERYELEMS; i++)
		sse += buf[i];

	hipMemcpy(buf, dev_err, QUERYELEMS * sizeof(double), hipMemcpyDeviceToHost);
	for (int i = 0; i < QUERYELEMS; i++)
		err_sum += buf[i];

	/* CALCULATE AND DISPLAY RESULTS */
	// these will be calculated on the CPU
	double mse = sse / QUERYELEMS;
	double ymean = compute_mean(query_ydata, QUERYELEMS);
	double var = compute_var(query_ydata, QUERYELEMS, ymean);
	double r2 = 1 - (mse / var);

	printf("Results for %d query points\n", QUERYELEMS);
	printf("APE = %.2f %%\n", err_sum / QUERYELEMS);
	printf("MSE = %.6f\n", mse);
	printf("R2 = 1 - (MSE/Var) = %.6lf\n", r2);

	printf("Total time = %lf secs\n", t_sum);
	// printf("Time for 1st query = %lf secs\n", t_first);
	// printf("Time for 2..N queries = %lf secs\n", t_sum - t_first);
	// printf("Average time/query = %lf secs\n", (t_sum - t_first) / (QUERYELEMS - 1));

	/* CLEANUP */

#if defined(DEBUG)
	/* Close the output file */
	fclose(fpout);
#endif

	free(mem);
	free(ydata);
	free(query_mem);
	free(query_ydata);
	// free(queries);
	free(buf);
        free(train_buf);
	free(query_buf);

        hipFree(dev_mem);
	hipFree(dev_ydata);
        hipFree(dev_query_ydata);
        // hipFree(dev_queries);

	hipFree(dev_sse);
	hipFree(dev_err);
	hipFree(dev_ntrain_blocks);
	hipFree(dev_train_block_size);

	return 0;
}
